#include "hip/hip_runtime.h"



#include <iostream>
#include <stdlib.h>
#include <stdio.h>

#include "../Utils/utils.h"
#include "../Utils/matutils.h"

#define PRINT 1

using namespace std;


__global__ void matmul1(double *a, double *b, double *c){

	//int tid = blockDim.y * threadIdx.y +   threadIdx.x;
	//int bid = gridDim.y * blockIdx.y + blockIdx.x; 

	//int idx = (blockDim.x * blockDim.y)*bid + tid;



	int idy = threadIdx.y + blockDim.y * blockIdx.y;
	int idx = threadIdx.x + blockDim.x * blockIdx.x;

//	printf("%d %d  \n", idy, idx);


	// int bid = (blockDim.x * blockDim.y) * ( gridblockIdx.y )

	//if(tid == 0){
	//	printf("bid %d   idx %d  \n ", bid, idx);
	//}
	//printf("tid %d/n", tid);


	int n = blockDim.x * gridDim.x;
	int k;
	double r = 0;
	for ( k=0; k< n  ; k++   ){
	
				 
		r +=  a[ n * idy + k  ] *  b[ n*k + idx  ];

	}

	c[ n * idy + idx ] = r;

//	 printf("%d %d - %f\n", idy, idx, r );
	
	



}



int main( int argc, char**  argv  ){

	int args_needed = 2;
	if (argc < args_needed + 1 ){
		printf(" Arg number error, needed: %d  \n", args_needed);
		return 0;	
	}


	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	printf(" CUDA - Maxmul  \n");


	// Size
	int n = atoi(argv[1]);
	int nt = atoi(argv[2]);

	//Create Data host n x n

	double *a;
	double *b;
	double *c;	

	a = (double *)malloc( sizeof(double) * n * n  );
	b = (double *)malloc( sizeof(double) * n * n  );
	c = (double *)malloc( sizeof(double) * n * n  );

	int i;

	for ( i =0; i<n*n ; i++  ){
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	
	print_dmatrix(a,n,n);
//	print_dmatrix(b,n,n);	

	// CUDA data
	double *a_dev;
	double *b_dev;
	double *c_dev;


	HANDLE_ERROR(  hipMalloc((void **)&a_dev, sizeof(double) * n * n)   );

	HANDLE_ERROR(  hipMalloc((void **)&b_dev, sizeof(double) * n * n)   );

	HANDLE_ERROR(  hipMalloc((void **)&c_dev, sizeof(double) * n * n)   );


	// Memcpy
	
	HANDLE_ERROR(  hipMemcpy(a_dev, a, sizeof(double) * n * n, hipMemcpyHostToDevice )     );
	HANDLE_ERROR(  hipMemcpy(b_dev, b, sizeof(double) * n * n, hipMemcpyHostToDevice )     );	
	
	
	// Kernel
	
	dim3 threads(nt, nt, 1);
	dim3 blocks(n/nt, n/nt, 1);

	
	hipEventRecord(start);
	matmul1<<< blocks, threads >>>(a_dev, b_dev, c_dev);
	hipEventRecord(stop);


	// Get data Devices
	HANDLE_ERROR(  hipMemcpy(c, c_dev, sizeof(double) * n * n, hipMemcpyDeviceToHost )     );
	

	hipEventSynchronize(stop);
	

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Time: %f\n", milliseconds );	

	print_dmatrix(c,n,n);

	return 0;
}






