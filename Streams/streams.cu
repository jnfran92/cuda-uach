#include "hip/hip_runtime.h"



#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include "../Utils/utils.h"
#include "../Utils/matutils.h"

#include "omp.h"

const int N = 1 << 20;


using namespace std;



__global__ void kernel(float *x, int n)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
		x[i] = sqrt(pow(3.14159,i));
	}
}



int main()
{


	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	const int num_streams = 8;

	hipStream_t streams[num_streams];
	float *data[num_streams];



	hipEventRecord(start);

	for (int i = 0; i < num_streams; i++) {
		hipStreamCreate(&streams[i]);

		hipMalloc(&data[i], N * sizeof(float));

		// launch one worker kernel per stream
		kernel<<<1, 64, 0, streams[i]>>>(data[i], N);

		// launch a dummy kernel on the default stream
		kernel<<<1, 1>>>(0, 0);
	}


	hipDeviceSynchronize();


	hipEventRecord(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("Time elapsed: %f   \n", milliseconds);




	hipDeviceReset();

	return 0;
}
