#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
/*#include <iostream>*/
#include "utils.h"

__global__ void kernel( int a, int b, int *c ){
	printf("Hello from CUDA Core\n");
	*c = a + b;
}


int main(int argc, char **argv ) {

	printf("Hello CUDA \n");
	



	int count_cuda_devs;
	HANDLE_ERROR(hipGetDeviceCount( &count_cuda_devs ));
	printf("Cuda-enabled devices =  %d  \n", count_cuda_devs);


	hipDeviceProp_t prop;
	
	int i;
	for (i = 0; i < count_cuda_devs; i++){

		HANDLE_ERROR(hipGetDeviceProperties( &prop, i ));
		printf("Device:%d\n	prop.name:  %s  \n",i,  prop.name);

		printf("	Memory: %zu  Mb \n", (prop.totalGlobalMem)/(1024*1024)   );
		
		printf( "	Clock rate: %d\n", prop.clockRate );

		printf("\n");

		printf("\n");

	}

	

	printf("Current device\n");
	int dev;
	HANDLE_ERROR( hipGetDevice( &dev ) );
    	printf( "ID of current CUDA device:  %d\n", dev );
	

	int c;
	int *dev_c;

	hipMalloc((void**)&dev_c, sizeof(int) );
	

	kernel<<<1,1>>>(999998,1,dev_c);


	hipMemcpy(&c, dev_c, sizeof(int) , hipMemcpyDeviceToHost  );

	printf("Res: %d  \n", c);

	hipFree(dev_c);	

	return 0;
}








