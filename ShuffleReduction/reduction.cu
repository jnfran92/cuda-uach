#include "hip/hip_runtime.h"



#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include "../Utils/utils.h"
#include "../Utils/matutils.h"

#include "omp.h"

/*#define BSIZE 2*/




using namespace std;


typedef long int92;




 __global__ void btree_reduction(  double *gvector, int92 n_steps, double *ovector ){
	

	__shared__ double vector[BSIZE];

	int92 i = threadIdx.y;
	int92 j = threadIdx.x;

	int92 gi = threadIdx.y + blockDim.y*blockIdx.y;
	int92 gj = threadIdx.x + blockDim.x*blockIdx.x;

	vector[j] = gvector[gj];
	__syncthreads();
	
	int92 count = 2;
	
	for (int92 m=0; m < n_steps; m++){
		
		
		if( j % count == 0  ){
			vector[j] += vector[j + count/2  ];
		}

		count = 2*count; 
		__syncthreads();
	}


	gvector[gj] = vector[j];
	
	if (j == 0){
		ovector[blockIdx.x] = vector[j]; 
	}



}



__global__ void atomic_add(double *vector){

	__shared__ double res;

	res = 0.0;	
	int92 i = threadIdx.y;
	int92 j = threadIdx.x;

	int92 gi = threadIdx.y + blockDim.y*blockIdx.y;
	int92 gj = threadIdx.x + blockDim.x*blockIdx.x;

	/*double res;i*/
	/*res = 0;*/
	__syncthreads();


	atomicAdd(&res,vector[j]  );

	__syncthreads();
	vector[0] = res;


}


__global__ void atomic_reduction(double *vector, double *gres){

	__shared__ double res;

	res = 0.0;	
	/*int92 i = threadIdx.y;*/
	int92 j = threadIdx.x;

	/*int92 gi = threadIdx.y + blockDim.y*blockIdx.y;*/
	int92 gj = threadIdx.x + blockDim.x*blockIdx.x;

	__syncthreads();

	atomicAdd(&res,vector[j]  );

	__syncthreads();


	if(j==0){
	atomicAdd(gres, res);
	}

}



__global__ void btree_atomic_reduction(double *gvector, int92 n_steps ,double *gres){

	__shared__ double vector[BSIZE];
	
	/*int92 i = threadIdx.y;*/
	int92 j = threadIdx.x;

	/*int92 gi = threadIdx.y + blockDim.y*blockIdx.y;*/
	int92 gj = threadIdx.x + blockDim.x*blockIdx.x;

	vector[j] = gvector[gj];
	
	
	__syncthreads();
	
	
	int92 count = 2;
	

	for (int92 m=0; m < n_steps; m++){
		
		if( j % count == 0  ){
			vector[j] += vector[j + count/2  ];
		}

		count = 2*count; 
		__syncthreads();
	}



	__syncthreads();


	if(j==0){
	atomicAdd(gres, vector[0]);
	}

}


__global__ void shuffle_reduction(double *gvector, int92 n_steps ,double *gres){

	/*__shared__ double vector[BSIZE];*/
	
	/*int92 i = threadIdx.y;*/
	int92 j = threadIdx.x;

	/*int92 gi = threadIdx.y + blockDim.y*blockIdx.y;*/
	int92 gj = threadIdx.x + blockDim.x*blockIdx.x;

	/*vector[j] = gvector[gj];*/


	double thread_var;
	
	thread_var = gvector[gj];

	
	__syncthreads();
	
	
	int92 count = BSIZE/2;
	

	for (int92 m=0; m < n_steps; m++){
		
		/*if( j % count == 0  ){*/
			/*vector[j] += vector[j + count/2  ];*/
		/*}*/

		thread_var += __shfl_down_sync(0xffffffff, thread_var, count, BSIZE);

		count = count/2; 
		/*__syncthreads();*/
	}



	__syncthreads();


	if(j==0){
	atomicAdd(gres, thread_var);
	}

}





void reduce(double *vector, int92 n   ){



	int92 n_steps = (int92)log2((double) n) ;
	/*printf("n_steps: %ld \n", n_steps );*/

	int92 n_temp = n/2;

	for(int92 i=0; i<n_steps; i++ ){



		for (int92 j=0; j<n_temp ; j++){
			int92 idx = j*(n/n_temp);
			vector[idx] = vector[idx] + vector[idx + ( n/(n_temp*2) )  ];
		}

		n_temp = n_temp/2;

	}



}



int main( int argc, char**  argv  ){

	int args_needed = 1;
	if (argc < args_needed + 1 ){
		printf(" Arg number error, needed: %d  \n", args_needed);
		return 0;
	}


	// Timers
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// OMP
	int ncpu = 1;
	omp_set_num_threads(ncpu);


	printf(" CUDA - Reduction \n");
	printf("BSIZE=%ld  \n",BSIZE );
	//Init parameters

	int92 n = atoi(argv[1]);

	int92 msize = sizeof(double)*n ;

	// Host Data

	double *a;
	double *b;
	/*double *c;*/

	printf("Request memory: %ld Mb   \n", msize/(1024*1024) );
	a = (double *) malloc( msize  );
	b = (double *) malloc( msize );

	// Create  Data
	for (int92 i=0; i<n; i++){

		a[i] = 1.0;
		b[i] = 0;
	}



	// CPU implementation--------------------------------


	float ms = 0.0;

	/*float t1=0.0, t2=0.0;*/

	/*print_dmatrix(a,1,n);*/


	hipEventRecord(start);
	reduce(a, n);
	hipEventRecord(stop);

	hipEventElapsedTime(&ms, start, stop);
	printf("%d GPU - Result: %f Time: %lf  \n", ncpu,a[0], ms );


	/*print_dmatrix(a,1,n);*/




	// GPU Implementation--------------------------------


	// Create  Data
	for (int92 i=0; i<n; i++){

		a[i] = 1.0;
		b[i] = 0;
	}



	// Device Data

	double *a_dev;
	double *b_dev;

	HANDLE_ERROR( hipMalloc((void **)&a_dev, msize)    );
	HANDLE_ERROR( hipMalloc((void **)&b_dev, msize)    );
		
	// Copy Data to Device
	HANDLE_ERROR( hipMemcpy(a_dev, a, msize  , hipMemcpyHostToDevice   )  );


	ms = 0.0;
	int92 n_steps = (int92)log2((double)BSIZE);
	/*printf("%ld  \n",n_steps);*/


	int92 gsize = n;

	dim3 block(BSIZE,1,1);
	dim3 grid(n/BSIZE,1,1);

	hipEventRecord(start);

	while(1){
		gsize= gsize/ BSIZE; 
		btree_reduction<<<gsize, block>>>(a_dev , n_steps, b_dev );
		hipDeviceSynchronize();
	
		/*printf("gsize: %ld \n",gsize);     */
		
		if(gsize < BSIZE){
			
			if(gsize>1){
			
				
				hipMemcpy(a_dev,b_dev, sizeof(double)*gsize , hipMemcpyDeviceToDevice);
				atomic_add<<<1, gsize>>>(a_dev);
				hipDeviceSynchronize();

			}
			
			break;
		}
		
		
		hipMemcpy(a_dev,b_dev, sizeof(double)*gsize , hipMemcpyDeviceToDevice);

	}
	
	hipEventRecord(stop);


	// Retrieve Data from Device
	// Get data Devices
	HANDLE_ERROR(  hipMemcpy(a, a_dev, msize, hipMemcpyDeviceToHost )     );

	HANDLE_ERROR(  hipMemcpy(b, b_dev, msize, hipMemcpyDeviceToHost )     );
	/*print_dmatrix(a,1,n);*/
	/*printf("Reduction result: %f\n", a[0]);*/

	ms = 0;
	hipEventElapsedTime(&ms, start, stop);
	printf("GPU Binary Tree %f, Time: %f \n",a[0] ,ms );
	



	// KERNEL 2

	// Create  Data
	for (int92 i=0; i<n; i++){

		a[i] = 1.0;
		/*b[i] = 0;*/
	}

	double *res_dev;	
	double res;

	res = 0.0;

	/*HANDLE_ERROR( hipMalloc((void **)&a_dev, msize)    );*/
	HANDLE_ERROR( hipMalloc((void **)&res_dev, sizeof(double))    );
		
	// Copy Data to Device
	HANDLE_ERROR( hipMemcpy(a_dev, a, msize  , hipMemcpyHostToDevice   )  );
	HANDLE_ERROR( hipMemcpy(res_dev, &res, sizeof(double)  , hipMemcpyHostToDevice   )  );
	
	ms = 0.0;

	dim3 block_atomic(BSIZE,1,1);
	dim3 grid_atomic(n/BSIZE,1,1);


	hipEventRecord(start);
	atomic_reduction<<<grid_atomic, block_atomic >>>(a_dev, res_dev );
	hipDeviceSynchronize();
	hipEventRecord(stop);
	
	
	HANDLE_ERROR( hipMemcpy(&res, res_dev, sizeof(double)  , hipMemcpyDeviceToHost   )  );



	ms = 0;
	hipEventElapsedTime(&ms, start, stop);
	printf("GPU Atomic Reduction %f, Time: %f \n",res ,ms );
	



	// KERNEL 3

	// Create  Data
	for (int92 i=0; i<n; i++){

		a[i] = 1.0;
		/*b[i] = 0;*/
	}

	/*double *res_dev;	*/
	/*double res;*/

	res = 0.0;

	/*HANDLE_ERROR( hipMalloc((void **)&a_dev, msize)    );*/
	/*HANDLE_ERROR( hipMalloc((void **)&res_dev, sizeof(double))    );*/
		
	// Copy Data to Device
	HANDLE_ERROR( hipMemcpy(a_dev, a, msize  , hipMemcpyHostToDevice   )  );
	HANDLE_ERROR( hipMemcpy(res_dev, &res, sizeof(double)  , hipMemcpyHostToDevice   )  );
	
	ms = 0.0;
	n_steps = (int92)log2((double)BSIZE);

	/*dim3 block_atomic(BSIZE,1,1);*/
	/*dim3 grid_atomic(n/BSIZE,1,1);*/


	hipEventRecord(start);
	btree_atomic_reduction<<<grid_atomic, block_atomic >>>(a_dev,n_steps, res_dev );
	hipDeviceSynchronize();
	hipEventRecord(stop);
	
	
	HANDLE_ERROR( hipMemcpy(&res, res_dev, sizeof(double)  , hipMemcpyDeviceToHost   )  );



	ms = 0;
	hipEventElapsedTime(&ms, start, stop);
	printf("GPU Btree Atomic Reduction %f, Time: %f \n",res ,ms );
	



	// KERNEL 4 SHUFFLE REDUCTION

	// Create  Data
	for (int92 i=0; i<n; i++){

		a[i] = 1.0;
		/*b[i] = 0;*/
	}

	/*double *res_dev;	*/
	/*double res;*/

	res = 0.0;

	/*HANDLE_ERROR( hipMalloc((void **)&a_dev, msize)    );*/
	/*HANDLE_ERROR( hipMalloc((void **)&res_dev, sizeof(double))    );*/
		
	// Copy Data to Device
	HANDLE_ERROR( hipMemcpy(a_dev, a, msize  , hipMemcpyHostToDevice   )  );
	HANDLE_ERROR( hipMemcpy(res_dev, &res, sizeof(double)  , hipMemcpyHostToDevice   )  );
	
	ms = 0.0;
	n_steps = (int92)log2((double)BSIZE);

	/*dim3 block_atomic(BSIZE,1,1);*/
	/*dim3 grid_atomic(n/BSIZE,1,1);*/


	hipEventRecord(start);
	shuffle_reduction<<<grid_atomic, block_atomic >>>(a_dev,n_steps, res_dev );
	hipDeviceSynchronize();
	hipEventRecord(stop);
	
	
	HANDLE_ERROR( hipMemcpy(&res, res_dev, sizeof(double)  , hipMemcpyDeviceToHost   )  );



	ms = 0;
	hipEventElapsedTime(&ms, start, stop);
	printf("GPU Shuffle Reduction %f, Time: %f \n",res ,ms );
	
	// Free memory

	hipFree( a_dev );
	hipFree( b_dev );

	free(a);
	free(b);





	return 0;
}





